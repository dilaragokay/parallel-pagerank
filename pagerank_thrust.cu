#include "hip/hip_runtime.h"
// compile with: nvcc -o a.out .\pagerank_thurst.cu -ccbin "C:\Program Files (x86)\Microsoft Visual Studio 14.0\VC\bin"
// run with: .\a.out

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <unordered_map>
#include <vector>
#include <iterator>
#include <ctime>
#include <memory>
#include <math.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <cmath>
#include <functional>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

using namespace std;

// CONSTANTS //
const string FILENAME = "graph.txt";
const double epsilon = pow(10, -6);
const double alpha = 0.2;
// CONSTANTS ENDS//

/*
  Returns absolute value of any type of value.
*/
template<typename T>
struct absolute_value : public unary_function<T,T>
{
  _host_ _device_ T operator()(const T &x) const
  {
    return x < T(0) ? -x : x;
  }
};

/*
  a: a constant
  x: a pointer
  Returns the result of  (a * x + (1 - a))
*/
struct saxpy_functor {
    const double a;
    saxpy_functor(double _a) : a(_a) {}

    _host_ _device_
        double operator()(const double& x) const {
            return a * x + (1 - a);
        }
};

void saxpy_fast(double A, thrust::device_vector<double>& X) {
    thrust::transform(X.begin(), X.end(), X.begin(), saxpy_functor(A));
}

int main() {
    ofstream myFile;
    myFile.open("output_thrust.csv");
    myFile << "Operation, Timing (s)" << endl;
    clock_t begin = clock();
    // main vectors of CSR format.
    thrust::host_vector<int> row_begin;     // row numbers of non-zero elements.
    thrust::host_vector<double> values;     // column numbers of non-zero elements.
    thrust::host_vector<int> col_indices;   // values of non-zero elements.

    // took word length of nodes.
    int word_length = 26;

    // read from file
    FILE *file;
    long size;
    char *buffer;
    size_t result;

    // read as a binary
    file = fopen(FILENAME.c_str(), "rb");
    if (file == NULL)
    {
        fputs("File Error", stderr);
        exit(1);
    }

    // go to end of the file
    fseek(file, 0, SEEK_END);

    // find the size of the file
    size = ftell(file);

    // go to start of the file
    rewind(file);

    buffer = (char *)malloc(sizeof(char) * size);
    if (buffer == NULL)
    {
        fputs("Memory Error", stderr);
        exit(2);
    }
    result = fread(buffer, 1, size, file);
    if (result != size)
    {
        fputs("Reading Error", stderr);
        exit(3);
    }

    cout << "file created " << endl;

    string s;
    s.assign(&buffer[size - (word_length + 1)], word_length);

    // umap of index for all Nodes
    unordered_map<string, int> input_list;
    // unorderd map for counters of all nodes.
    unordered_map<int, int> counter_list;

    string old = "";
    int count = 0;

    string a, b;
    int index = 0;

    /*
    read all lines and numbered only outgoing sides to edges.
    fill the row_begin vector.
    */
    for (int i = 0; i < size; i += ((word_length + 1) * 2))
    {
        b.assign(&buffer[i + (word_length + 1)], word_length);
        unordered_map<string, int>::iterator it_b = input_list.find(b);

        if (it_b == input_list.end())
        {
            input_list.insert(make_pair(b, index));
            counter_list.insert(make_pair(index, 0));
            index++;
        }
        if (old != b)
        {
            row_begin.push_back(count);
        }
        old = b;
        count++;
    }
    row_begin.push_back(count);

    /*
    read all lines and numbered necessery nodes.
    fill the col_indices vector.
    */
    for (int i = 0; i < size; i += ((word_length + 1) * 2))
    {
        a.assign(&buffer[i], word_length);
        unordered_map<string, int>::iterator it_a = input_list.find(a);

        if (it_a == input_list.end())
        {
            input_list.insert(make_pair(a, index));
            counter_list.insert(make_pair(index, 1));
            row_begin.push_back(count);
            index++;
        }
        else
        {
            unordered_map<int, int>::iterator it_a_counter = counter_list.find(it_a->second);
            it_a_counter->second++;
        }
        it_a = input_list.find(a);
        col_indices.push_back(it_a->second);
    }

    /*
    fill values vector.
    */
    int k = 0;
    for (int i = 1; i < row_begin.size(); i++)
    {
        for (int j = row_begin[i - 1]; j < row_begin[i]; j++)
        {
            unordered_map<int, int>::iterator it_counter = counter_list.find(col_indices[k]);
            if (it_counter != counter_list.end()) {
              values.push_back((double)1 / (it_counter->second));
            }
            k++;
        }
    }

    int M = row_begin.size() - 1;  // number of nodes
    clock_t end = clock();
    double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    myFile << "I/O, " << elapsed_secs << endl;

    // initial vectors for multiplication
    thrust::host_vector<double> r_t_host(row_begin.size() - 1, 1.0);    // 1, 1, 1, 1, 1...
    thrust::host_vector<double> r_t_1_host(row_begin.size() - 1, 1.0);  // 1, 1, 1, 1, 1...

    // row_ctr will be multiply with values
    thrust::host_vector<int> row_ctr(values.size());
    for (int line = 1; line < row_begin.size(); line++) {
      for (int i = row_begin[line - 1]; i < row_begin[line]; i++) {
        row_ctr[i] = line;   // filled with row numbers
      }
    }

    double norm = pow(10, -6);    // initial value of norm is equal to epsilon
    begin = clock();
    // while L1 norm of r_t vector is larger than epsilon
    // Matrix multiplication is done in two parts
    // 1. multiplying corresponding elements one by one
    // 2. summing the multiplied elements calculated in 1
    while (norm >= epsilon) {
      thrust::device_vector<double> r_t = r_t_host;
      thrust::device_vector<double> r_t_1 = r_t_1_host;

      r_t = r_t_1;
      thrust::host_vector<double> r_t_ = r_t;
      thrust::host_vector<double> mult_vec(col_indices.size());

      // Write values of r_t which correspond to a column
      for (int i = 0; i < col_indices.size(); i++) {
        mult_vec[i] = r_t_[col_indices[i]];
      }

      thrust::device_vector<double> mult_vec_new = mult_vec;
      thrust::device_vector<double> reisss(values.size());
      thrust::multiplies<double> multOp;

      thrust::device_vector<double> val_device = values;
      // Multiply val_device with mult_vec_new elementwise and write the result to reisss
      thrust::transform(val_device.begin(), val_device.end(), mult_vec_new.begin(), reisss.begin(), multOp );

      thrust::device_vector<int> row_device = row_begin;
      thrust::device_vector<int> row_ctr_new = row_ctr;

      // row_ctr_new has the same size of values. For each value in values, the
      // element in the row_ctr_new with the same index is the row number of the
      // value. reduce_by_key sums the values in reisss which are in the same
      // row. Namely, it reducesrow_ctr_new to new r_t_1 vector which has the
      // final matrix multiplication results
      thrust::reduce_by_key(row_ctr_new.begin(), row_ctr_new.end(), reisss.begin(), row_device.begin(), r_t_1.begin());

      saxpy_fast(alpha, r_t_1);

      thrust::minus<double> op2;
      thrust::transform(r_t_1.begin(), r_t_1.end(), r_t.begin(), r_t.begin(), op2 );

      thrust::transform(r_t.begin(), r_t.end(), r_t.begin(), absolute_value<double>());

      norm = thrust::reduce(r_t.begin(), r_t.end());

      r_t_host = r_t;
      r_t_1_host = r_t_1;
    }
    end = clock();
    elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    myFile << "PageRank, " << elapsed_secs << endl;

    thrust::host_vector<double> first_5 = r_t_1_host;
    // took maximum 5 ranks and second array took the indexes of them.
    int arr[5] = {0, 0, 0, 0, 0};
    int arr_index[5] = {0, 0, 0, 0, 0};

    for (int i = 0; i < first_5.size(); i++) {
        if (first_5[i] > arr[0]) {
            arr[4] = arr[3];
            arr_index[4] = arr_index[3];
            arr[3] = arr[2];
            arr_index[3] = arr_index[2];
            arr[2] = arr[1];
            arr_index[2] = arr_index[1];
            arr[1] = arr[0];
            arr_index[1] = arr_index[0];
            arr[0] = first_5[i];
            arr_index[0] = i;
        } else if (first_5[i] > arr[1]) {
            arr[4] = arr[3];
            arr_index[4] = arr_index[3];
            arr[3] = arr[2];
            arr_index[3] = arr_index[2];
            arr[2] = arr[1];
            arr_index[2] = arr_index[1];
            arr[1] = first_5[i];
            arr_index[1] = i;
        } else if (first_5[i] > arr[2]) {
            arr[4] = arr[3];
            arr_index[4] = arr_index[3];
            arr[3] = arr[2];
            arr_index[3] = arr_index[2];
            arr[2] = first_5[i];
            arr_index[2] = i;
        } else if (first_5[i] > arr[3]) {
            arr[4] = arr[3];
            arr_index[4] = arr_index[3];
            arr[3] = first_5[i];
            arr_index[3] = i;
        } else if (first_5[i] > arr[4]) {
            arr[4] = first_5[i];
            arr_index[4] = i;
        }
    }
    myFile << "Top 5 hosts," << endl;
    // print top 5 ranked strings.
    for (int k = 0; k < 5; k++) {
        for ( unordered_map<string, int>::iterator it_counter = input_list.begin(); it_counter != input_list.end(); ++it_counter ) {
            if (it_counter->second == arr_index[k]) {
                myFile << it_counter->first << "," << endl;
            }
        }
    }
    myFile.close();
    return 0;
}
